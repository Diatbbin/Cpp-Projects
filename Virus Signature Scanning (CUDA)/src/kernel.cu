#include "hip/hip_runtime.h"
#include <vector>
#include "defs.h"
#include <iostream>


__global__ void matchFile(const uint8_t* file_data, size_t file_len, char** signatures, size_t* lens, int* founds)
{
    // TODO: your code!
    size_t sig_len = lens[blockIdx.x]/2;
    char* signature = signatures[blockIdx.x];

    size_t start = threadIdx.x;
    size_t end = file_len - sig_len + 1;
    size_t stride = blockDim.x;


    for (size_t i = start; i < end; i += stride)
    {
        size_t j;
        for (j = 0; j != sig_len; ++j)
        {
            const char h1 = signature[j * 2];
            const char h2 = signature[j * 2 + 1];
            uint8_t b1;
            uint8_t b2;
            if (h1 >= '0' && h1<= '9')
                b1 = h1 - '0';
            else if (h1 == '?')
                b1 = file_data[i + j] >> 4;
            else
                b1 = h1 - 'a' + 10;

            if (h2 >= '0' && h2 <= '9')
                b2 = h2 - '0';
            else if (h2 == '?')
                b2 = file_data[i + j] & 15;
            else
                b2 = h2 - 'a' + 10;

            uint8_t byte = (b1 << 4) | b2;

            if (file_data[i + j] != byte) break;
        }

        if (j == sig_len)
            atomicExch(&founds[blockIdx.x], 1);
        
        if (founds[blockIdx.x] == 1) return;

    }
}

void runScanner(std::vector<Signature>& signatures, std::vector<InputFile>& inputs)
{
    {
        hipDeviceProp_t prop;
        check_cuda_error(hipGetDeviceProperties(&prop, 0));

        fprintf(stderr, "cuda stats:\n");
        fprintf(stderr, "  # of SMs: %d\n", prop.multiProcessorCount);
        fprintf(stderr, "  global memory: %.2f MB\n", prop.totalGlobalMem / 1024.0 / 1024.0);
        fprintf(stderr, "  shared mem per block: %zu bytes\n", prop.sharedMemPerBlock);
        fprintf(stderr, "  constant memory: %zu bytes\n", prop.totalConstMem);
    }

    /*
       Here, we are creating one stream per file just for demonstration purposes;
       you should change this to fit your own algorithm and/or implementation.
     */
    std::vector<hipStream_t> streams {};
    streams.resize(inputs.size());

    std::vector<uint8_t*> file_bufs {};
    std::vector<size_t*> file_size_bufs {};
    for(size_t i = 0; i < inputs.size(); i++)
    {
        hipStreamCreate(&streams[i]);

        // allocate memory on the device for the file
        uint8_t* ptr = 0;
        check_cuda_error(hipMalloc(&ptr, inputs[i].size));
        file_bufs.push_back(ptr);
    }

    // allocate memory for the signatures
    std::vector<char*> sig_bufs {};
    size_t* d_sig_size;
    size_t* h_sig_size = (size_t*)malloc(sizeof(size_t) * signatures.size());
    size_t size = signatures.size() * sizeof(size_t);
    check_cuda_error(hipMalloc((void**) &d_sig_size, size));

    char** d_sig_bufs;
    char** h_sig_bufs = (char**)malloc(sizeof(char*) * signatures.size());
    size_t size_data = signatures.size() * sizeof(char*);
    check_cuda_error(hipMalloc((void**) &d_sig_bufs, size_data));

    for(size_t i = 0; i < signatures.size(); i++)
    { 
        char* ptr = 0;
        h_sig_size[i] = signatures[i].size;
        check_cuda_error(hipMalloc(&ptr, signatures[i].size));
        hipMemcpy(ptr, signatures[i].data, signatures[i].size, hipMemcpyHostToDevice);
        h_sig_bufs[i] = ptr;
    }
    hipMemcpy(d_sig_size, h_sig_size, size, hipMemcpyHostToDevice);
    hipMemcpy(d_sig_bufs, h_sig_bufs, size_data, hipMemcpyHostToDevice);
    // allocate memory for the results

    std::vector<int*> results;
    for (int i = 0; i != inputs.size(); ++i)
    {
        int* ptr = 0;
        hipMallocManaged(&ptr, sizeof(int) * signatures.size());
        hipMemset(ptr, 0, sizeof(int) * signatures.size());
        results.push_back(ptr);
    }

    for(size_t file_idx = 0; file_idx < inputs.size(); file_idx++)
    {
        // asynchronously copy the file contents from host memory
        // (the `inputs`) to device memory (file_bufs, which we allocated above)
        hipMemcpyAsync(file_bufs[file_idx], inputs[file_idx].data, inputs[file_idx].size,
                hipMemcpyHostToDevice, streams[file_idx]);    // pass in the stream here to do this async

            // launch the kernel!
            // your job: figure out the optimal dimensions
    
            /*
               This launch happens asynchronously. This means that the CUDA driver returns control
               to our code immediately, without waiting for the kernel to finish. We can then
               run another iteration of this loop to launch more kernels.

               Each operation on a given stream is serialised; in our example here, we launch
               all signatures on the same stream for a file, meaning that, in practice, we get
               a maximum of NUM_INPUTS kernels running concurrently.

               Of course, the hardware can have lower limits; on Compute Capability 8.0, at most
               128 kernels can run concurrently --- subject to resource constraints. This means
               you should *definitely* be doing more work per kernel than in our example!
             */
        matchFile<<<signatures.size(), 512, /* shared memory per block: */ 0, streams[file_idx]>>>(
                file_bufs[file_idx], inputs[file_idx].size,
                d_sig_bufs, d_sig_size, results[file_idx]);


            // example output printing. don't forget to change this!
            // printf("%s: %s\n", inputs[file_idx].name.c_str(), signatures[sig_idx].name.c_str());
    }

    hipDeviceSynchronize();

    // print the results

    for (int i = 0; i != inputs.size(); ++i)
    {
        for (int j = 0; j != signatures.size(); ++j)
        {
            if (results[i][j] == 1) printf("%s: %s\n", inputs[i].name.c_str(), signatures[j].name.c_str());
        }
    }

    // free the device memory, though this is not strictly necessary
    // (the CUDA driver will clean up when your program exits)
    for(auto buf : file_bufs)
        hipFree(buf);

    for(auto buf : sig_bufs)
        hipFree(buf);

    for (auto buf: results)
        hipFree(buf);

    // clean up streams (again, not strictly necessary)
    for(auto& s : streams)
        hipStreamDestroy(s);
}
